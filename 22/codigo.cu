#include <iostream>
// #include <vector> // Não precisamos mais deste
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// --- Definições Globais da Simulação ---
#define NX 128
#define NY 128
#define NZ 128
#define STEPS 100

// Constantes físicas
#define DX 0.01
#define DY 0.01
#define DZ 0.01
#define DT 0.0001
#define NU 0.1

// --- Kernel CUDA (sem alterações) ---
__global__ void atualiza(double *vnew, double *vold,
                         int nx, int ny, int nz, double alpha){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x > 0 && x < nx - 1 && y > 0 && y < ny - 1 && z > 0 && z < nz - 1){
    int idx = z * ny * nx + y * nx + x;
    int xm = idx - 1;
    int xp = idx + 1;
    int ym = idx - nx;
    int yp = idx + nx;
    int zm = idx - nx * ny;
    int zp = idx + nx * ny;

    vnew[idx] = vold[idx] + alpha * (vold[xp] + vold[xm] +
                                     vold[yp] + vold[ym] +
                                     vold[zp] + vold[zm] - 6.0 * vold[idx]);
  }
}

double calcular_soma_centro(const double *u, int nx, int ny, int nz, int tamanho_bloco){
  // Calcula as coordenadas de início do cubo central
  int start_x = (nx / 2) - (tamanho_bloco / 2);
  int start_y = (ny / 2) - (tamanho_bloco / 2);
  int start_z = (nz / 2) - (tamanho_bloco / 2);

  double soma = 0.0;

  // Loop triplo para percorrer o cubo central
  for (int z = start_z; z < start_z + tamanho_bloco; z++){
    for (int y = start_y; y < start_y + tamanho_bloco; y++){
      for (int x = start_x; x < start_x + tamanho_bloco; x++){
        long long idx = (long long)z * ny * nx + (long long)y * nx + x;
        soma += u[idx];
      }
    }
  }
  return soma;
}

// --- Programa Principal (Host) ---
int main(){
  // --- 1. Configuração Inicial ---
  const int nx = NX, ny = NY, nz = NZ;
  const int nt = STEPS;
  printf("Iniciando simulação em GPU com CUDA (usando malloc/free)\n");
  printf("Grade: %d x %d x %d, Passos: %d\n", nx, ny, nz, nt);

  const double alpha = NU * DT / (DX * DX);
  const size_t num_elements = (size_t) nx * ny * nz;
  const size_t size_bytes = num_elements * sizeof(double);

  // --- 2. Alocação de Memória no Host (CPU) com malloc/calloc ---
  double *h_vold, *h_vfinal;
  // Usamos calloc para h_vold para já inicializar com zeros
  h_vold = (double *)calloc(num_elements, sizeof(double));
  h_vfinal = (double *)malloc(size_bytes);

  // Verificação de erro na alocação
  if (h_vold == NULL || h_vfinal == NULL){
    fprintf(stderr, "Erro: Falha na alocação de memória do host.\n");
    free(h_vold); // Libera o que possa ter sido alocado
    free(h_vfinal);
    return 1;
  }

  // --- 3. Definindo a Condição Inicial ---
  printf("Definindo condição inicial no host...\n");
  // Como usamos calloc, h_vold já está preenchido com zeros.
  // Apenas definimos a perturbação central.
  int cx = nx / 2, cy = ny / 2, cz = nz / 2;
  h_vold[cz * ny * nx + cy * nx + cx] = 1.0;

  // --- 4. Alocação de Memória na GPU (Device) ---
  printf("Alocando memória na GPU...\n");
  double *d_vold, *d_vnew;
  hipMalloc(&d_vold, size_bytes);
  hipMalloc(&d_vnew, size_bytes);

  // --- 5. Cópia dos Dados Iniciais para a GPU ---
  hipMemcpy(d_vold, h_vold, size_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_vnew, h_vold, size_bytes, hipMemcpyHostToDevice);

  // --- 6. Definindo a Grade de Execução CUDA ---
  const int block_dim_x = 8;
  const int block_dim_y = 8;
  const int block_dim_z = 8;

  dim3 threadsPerBlock(block_dim_x, block_dim_y, block_dim_z);
  dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                 (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);
  printf("Configuração CUDA: %d blocos, %d threads/bloco\n", numBlocks.x * numBlocks.y * numBlocks.z, threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z);

  // --- 7. Medição de Tempo e Execução do Loop ---
  printf("Iniciando loop de simulação na GPU...\n");
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (int t = 0; t < nt; t++){
    atualiza<<<numBlocks, threadsPerBlock>>>(d_vnew, d_vold, nx, ny, nz, alpha);
    double *tmp = d_vold;
    d_vold = d_vnew;
    d_vnew = tmp;
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;

  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Simulação na GPU concluída.\n");
  printf("Tempo de execução: %.4f ms (%.6f s)\n", milliseconds, milliseconds / 1000.0);

  // --- 8. Cópia dos Resultados de Volta para o Host ---
  printf("Copiando resultados de volta para o host...\n");
  hipMemcpy(h_vfinal, d_vold, size_bytes, hipMemcpyDeviceToHost);

  // --- CÁLCULO DE VERIFICAÇÃO ---
  printf("Calculando soma de verificação em um bloco de %dx%dx%d...\n", block_dim_x, block_dim_y, block_dim_z);
  // Passamos a dimensão do bloco (usamos a do eixo X, assumindo que são iguais)
  double soma_gpu = calcular_soma_centro(h_vfinal, nx, ny, nz, block_dim_x);
  printf("----------------------------------------\n");
  printf("SOMA DE VERIFICAÇÃO (GPU): %.15f\n", soma_gpu);
  printf("----------------------------------------\n");

  // --- 9. Liberação de Recursos ---
  printf("Liberando memória.\n");
  
  // Libera a memória da GPU
  hipFree(d_vold);
  hipFree(d_vnew);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Libera a memória do Host
  free(h_vold);
  free(h_vfinal);

  return 0;
}