#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// --- Definições Globais da Simulação ---
#define NX 128
#define NY 128
#define NZ 128
#define STEPS 100

// Constantes físicas
#define DX 0.01
#define DY 0.01
#define DZ 0.01
#define DT 0.0001
#define NU 0.1

// --- NOVAS CONSTANTES PARA A OTIMIZAÇÃO ---
#define BLOCK_DIM 8 // Dimensão do nosso bloco de threads (8x8x8)
#define RADIUS 1    // Raio do stencil (1 para vizinhos diretos)

// --- KERNEL ANTIGO REMOVIDO ---
// O kernel __global__ void atualiza(...) foi substituído pelo abaixo.

// --- NOVO KERNEL OTIMIZADO COM MEMÓRIA COMPARTILHADA ---
__global__ void atualiza_shared(double *vnew, double *vold,
                                int nx, int ny, int nz, double alpha){
  // --- Declaração da Memória Compartilhada ("Bancada de Trabalho" 3D) ---
  // Um cubo de (8+2*1) x (8+2*1) x (8+2*1) = 10x10x10
  __shared__ double tile[BLOCK_DIM + 2 * RADIUS][BLOCK_DIM + 2 * RADIUS][BLOCK_DIM + 2 * RADIUS];

  // Índices da thread dentro do bloco (0 a 7)
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

  // Índices globais do ponto que esta thread irá calcular
  int gx = blockIdx.x * BLOCK_DIM + tx;
  int gy = blockIdx.y * BLOCK_DIM + ty;
  int gz = blockIdx.z * BLOCK_DIM + tz;
  long long gidx = (long long) gz * ny * nx + (long long) gy * nx + gx;

  // Índices para escrever na memória compartilhada (com a borda/halo)
  int tile_x = tx + RADIUS;
  int tile_y = ty + RADIUS;
  int tile_z = tz + RADIUS;

  // --- Fase 1: Carregar dados da memória GLOBAL para a COMPARTILHADA ---
  // Cada thread carrega seu ponto principal para o centro do "tile"
  if (gx < nx && gy < ny && gz < nz){
    tile[tile_z][tile_y][tile_x] = vold[gidx];
  }

  // Threads nas bordas do bloco carregam o "halo"
  // Halo no eixo X
  if (tx < RADIUS && gx >= RADIUS)
    tile[tile_z][tile_y][tile_x - RADIUS] = vold[gidx - RADIUS];
  if (tx >= BLOCK_DIM - RADIUS && gx + RADIUS < nx)
    tile[tile_z][tile_y][tile_x + RADIUS] = vold[gidx + RADIUS];

  // Halo no eixo Y
  if (ty < RADIUS && gy >= RADIUS)
    tile[tile_z][tile_y - RADIUS][tile_x] = vold[gidx - nx];
  if (ty >= BLOCK_DIM - RADIUS && gy + RADIUS < ny)
    tile[tile_z][tile_y + RADIUS][tile_x] = vold[gidx + nx];

  // Halo no eixo Z
  if (tz < RADIUS && gz >= RADIUS)
    tile[tile_z - RADIUS][tile_y][tile_x] = vold[gidx - (long long)nx * ny];
  if (tz >= BLOCK_DIM - RADIUS && gz + RADIUS < nz)
    tile[tile_z + RADIUS][tile_y][tile_x] = vold[gidx + (long long)nx * ny];

  // --- Fase 2: Sincronizar todas as threads do bloco ---
  __syncthreads();

  // --- Fase 3: Calcular usando dados da memória COMPARTILHADA ---
  if (gx > 0 && gx < nx - 1 && gy > 0 && gy < ny - 1 && gz > 0 && gz < nz - 1){
    double center_val = tile[tile_z][tile_y][tile_x];

    double sum_neighbors = tile[tile_z][tile_y][tile_x + 1] + tile[tile_z][tile_y][tile_x - 1] +
                           tile[tile_z][tile_y + 1][tile_x] + tile[tile_z][tile_y - 1][tile_x] +
                           tile[tile_z + 1][tile_y][tile_x] + tile[tile_z - 1][tile_y][tile_x];

    vnew[gidx] = center_val + alpha * (sum_neighbors - 6.0 * center_val);
  }
}

// Função de validação (sem alterações)
double calcular_soma_centro(const double *u, int nx, int ny, int nz, int tamanho_bloco){
  int start_x = (nx / 2) - (tamanho_bloco / 2);
  int start_y = (ny / 2) - (tamanho_bloco / 2);
  int start_z = (nz / 2) - (tamanho_bloco / 2);
  double soma = 0.0;
  for (int z = start_z; z < start_z + tamanho_bloco; z++)
  {
    for (int y = start_y; y < start_y + tamanho_bloco; y++)
    {
      for (int x = start_x; x < start_x + tamanho_bloco; x++)
      {
        long long idx = (long long)z * ny * nx + (long long)y * nx + x;
        soma += u[idx];
      }
    }
  }
  return soma;
}

// --- Programa Principal (Host) ---
int main(){
  // --- 1. Configuração Inicial (sem alterações) ---
  const int nx = NX, ny = NY, nz = NZ;
  const int nt = STEPS;
  printf("Iniciando simulação em GPU com CUDA (OTIMIZADO com __shared__)\n"); // Título atualizado
  printf("Grade: %d x %d x %d, Passos: %d\n", nx, ny, nz, nt);
  const double alpha = NU * DT / (DX * DX);
  const size_t num_elements = (size_t)nx * ny * nz;
  const size_t size_bytes = num_elements * sizeof(double);

  // --- 2. Alocação de Memória no Host (sem alterações) ---
  double *h_vold, *h_vfinal;
  h_vold = (double *)calloc(num_elements, sizeof(double));
  h_vfinal = (double *)malloc(size_bytes);
  if (h_vold == NULL || h_vfinal == NULL)
  {
    fprintf(stderr, "Erro: Falha na alocação de memória do host.\n");
    free(h_vold);
    free(h_vfinal);
    return 1;
  }

  // --- 3. Condição Inicial (sem alterações) ---
  printf("Definindo condição inicial no host...\n");
  int cx = nx / 2, cy = ny / 2, cz = nz / 2;
  h_vold[cz * ny * nx + cy * nx + cx] = 1.0;

  // --- 4. Alocação de Memória na GPU (sem alterações) ---
  printf("Alocando memória na GPU...\n");
  double *d_vold, *d_vnew;
  hipMalloc(&d_vold, size_bytes);
  hipMalloc(&d_vnew, size_bytes);

  // --- 5. Cópia dos Dados Iniciais para a GPU (sem alterações) ---
  hipMemcpy(d_vold, h_vold, size_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_vnew, h_vold, size_bytes, hipMemcpyHostToDevice);

  // --- 6. Definindo a Grade de Execução CUDA ---
  // MUDANÇA: Usando a macro para definir o tamanho do bloco
  dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM, BLOCK_DIM);
  dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                 (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);
  printf("Configuração CUDA: %d blocos, %d threads/bloco\n", numBlocks.x * numBlocks.y * numBlocks.z, threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z);

  // --- 7. Medição de Tempo e Execução do Loop ---
  printf("Iniciando loop de simulação na GPU...\n");
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (int t = 0; t < nt; t++)
  {
    // MUDANÇA: Chamando o novo kernel otimizado
    atualiza_shared<<<numBlocks, threadsPerBlock>>>(d_vnew, d_vold, nx, ny, nz, alpha);
    double *tmp = d_vold;
    d_vold = d_vnew;
    d_vnew = tmp;
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Simulação na GPU concluída.\n");
  printf("Tempo de execução: %.4f ms (%.6f s)\n", milliseconds, milliseconds / 1000.0);

  // --- 8. Cópia dos Resultados de Volta para o Host (sem alterações) ---
  printf("Copiando resultados de volta para o host...\n");
  hipMemcpy(h_vfinal, d_vold, size_bytes, hipMemcpyDeviceToHost);

  // --- CÁLCULO DE VERIFICAÇÃO ---
  // MUDANÇA: Usando a macro para consistência no tamanho do bloco de verificação
  printf("Calculando soma de verificação em um bloco de %dx%dx%d...\n", BLOCK_DIM, BLOCK_DIM, BLOCK_DIM);
  double soma_gpu = calcular_soma_centro(h_vfinal, nx, ny, nz, BLOCK_DIM);
  printf("----------------------------------------\n");
  printf("SOMA DE VERIFICAÇÃO (GPU): %.15f\n", soma_gpu);
  printf("----------------------------------------\n");

  // --- 9. Liberação de Recursos (sem alterações) ---
  printf("Liberando memória.\n");
  hipFree(d_vold);
  hipFree(d_vnew);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  free(h_vold);
  free(h_vfinal);

  return 0;
}